#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "Binary_operations.h"


__global__ void kernelEnumerate(int n, int *matr, int *sols, int start_sol)
{
    int blockId = blockIdx.x * gridDim.y + blockIdx.y;
    int idx = blockId * blockDim.x  +  threadIdx.x;
	int sol = idx + start_sol;
	
	int c = 0;
	for(int i=0; i<n; i++)
	{
		if(GetBit(sol,i))
		{	c++;
			for(int j=0; j<i; j++)
			{
				if(GetBit(sol,j))
				{
					if(matr[i*n + j])
					{
					    sols[idx] = -1;
						return ;
					}
				}			
			}
		}		
	}	
	sols[idx] = c;
}



// find maximal element in a small part of the array
__global__ void kernelFindMax(int part, int* sols, int* devMaxArray, int* devMaxindArray)
{
    int blockId = blockIdx.x;
    int id = blockId * blockDim.x  +  threadIdx.x;
	
	int max=0, imax;
	
	for(int i=0; i<part; i++)
	{
	   if(sols[i + part*id] > max)
	   {
	      max = sols[i + part*id];
	      imax = i + part*id;
	   }
	}
	devMaxArray[id] = max;
	devMaxindArray[id] = imax;
}



void findMax(int n, int *maxArray, int *maxIndArray, int &max, int &maxind)
{
   int i;
   max=0;
   for(int i=0;i<n;i++)
   {
      if(maxArray[i] > max)
      {
         max = maxArray[i];
         maxind = maxIndArray[i];
      }
   }
}



int solve(int n, int **matr)
{

    int N = 1<<n;
    int i,j,k;
   
    int *matr1 = new int [n*n];
    k=0;
    for(i = 0; i < n; i++) {
        for(j = 0; j < n; j++) {
            matr1[k] = matr[i][j];
            k++;
        }
    }
    int *devMatr;
    hipMalloc((void**)&devMatr, n * n * sizeof(int) );
    hipMemcpy(devMatr,    matr1,   n * n * sizeof(int), hipMemcpyHostToDevice);

    int blocksX=128;
    int blocksY=128;
    int threadsX=32;
    int partSize = blocksX*blocksY*threadsX;

    int *devSols;
    hipMalloc((void**)&devSols, partSize * sizeof(int) );

    int *sols = new int[partSize];
    for(i=0; i<partSize; i++)
       sols[i]=0;


    // define parts for the REDUCTION step for finding the maximal value in array
    int numSmallParts = 8*128;  //=1024
    int smallPartSize = partSize / numSmallParts; 

    int *devMaxArray, *devMaxIndArray;
    hipMalloc((void**)&devMaxArray, numSmallParts * sizeof(int) );
    hipMalloc((void**)&devMaxIndArray, numSmallParts * sizeof(int) );

    int *maxArray = new int[numSmallParts],
        *maxIndArray = new int[numSmallParts];


    int start_sol = 0, part = 0;
    int global_max = 0;
    do
    {
      dim3 blocks  = dim3(blocksX, blocksY);
      dim3 threads = dim3(threadsX);

      //enumerate all the solutions in the current part
      kernelEnumerate<<<blocks,threads>>>(n, devMatr, devSols, start_sol);
      start_sol += partSize;

      // reductio step: fird maximums in small portions of the array and store them in a new array
      blocks  = dim3(8);
      threads = dim3(128);
      kernelFindMax<<<blocks,threads>>>(smallPartSize, devSols, devMaxArray, devMaxIndArray);
      hipMemcpy(maxArray, devMaxArray,  numSmallParts * sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(maxIndArray, devMaxIndArray,  numSmallParts * sizeof(int), hipMemcpyDeviceToHost);

      int max, maxind;
      // find maximal independent set in the curent part
      findMax(numSmallParts, maxArray, maxIndArray, max, maxind);
      printf("part = %i,  max=%i \n", part, max);
      part ++;
      if (global_max < max)
          global_max = max;

    } while(start_sol < N);

return global_max;
}

